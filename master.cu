#include "GraphManager.h"
#include <string>
#include <iostream>
#include "kernels.h"
#include <sys/time.h>
#include <sys/stat.h>

int askOptions() {
	printf("\nEnter <1> For Gupta Kulkarni.\nEnter <2> for Hedetniemi et al.\nEnter <3> for Turau.\nEnter <4> for lock generic.\nEnter <0> to exit the program.\nEnter your choice : ");
	int op = 0;
	scanf("%d", &op);
	std::cout<< "\n\n";
	
	return op;
}

int getGraphValueForFinalCheck(int* g, int row, int col) {
	int start = g[row + 1];
	return g[start + col];
}

bool finalCheckMaximallyIndependent(int* g, int *states, int *degrees, int order) {
	for(int i = 0; i < order; i++) {
		if(states[i] == 1){
			for(int jLoop = 0; jLoop < degrees[i]; jLoop++){
				int j = getGraphValueForFinalCheck(g, i, jLoop);
				if(states[j] != 0) return false;
			}
		} else {
			if(states[i] != 0) return false;
			
			bool allOut = true;
			for(int jLoop = 0; jLoop < degrees[i]; jLoop++){
				int j = getGraphValueForFinalCheck(g, i, jLoop);
				if(states[j] == 1) allOut = false;
			}
			if(allOut == true) return false;
		}
	}
	return true;
}

long int arraySum(int* a, int size){
	long int sum = 0;
	for(int i = 0; i < size; i++){
		sum += a[i];
	}
	return sum;
}

bool driver(int* d, int* g, int *s, int order, int size){
	int* miscbool = (int*)malloc(4 * sizeof(int)); miscbool[0] = 0; miscbool[1] = 0; miscbool[2] = 0; miscbool[3] = 0;
	//problem?, stop?, lockstep?, lockStepLevel
	
	int* p = (int*)malloc(order * sizeof(int)); // pointers
	int* a1 = (int*)malloc(order * sizeof(int)); //isCurrentlyExecuting
	int* a2 = (int*)malloc(order * sizeof(int)); //timesExecuted
	int* a3 = (int*)malloc(order * sizeof(int)); //wantToExecute
	int* a4 = (int*)malloc(order * sizeof(int)); //stateChanged
	int* a5 = (int*)malloc(order * sizeof(int)); //stateNotChanged
	int* a6 = (int*)malloc(order * sizeof(int)); //lockstepNumber
	
	int op = askOptions();
	if(op <= 0 || op >= 5) {
		printf("Program exiting.\n");
		return true;
	}
	
	for(int i=0; i<order; i++) {
		s[i] = 1;
		p[i] = -1;
		a1[i] = -1;
		a2[i] = -1;
		a3[i] = -1;
		a4[i] = 0;
		a5[i] = 0;
		a6[i] = 0;
	}
	
	int* cores = (int*)malloc(sizeof(int)); cores[0] = 0;
	int* moves = (int*)malloc(sizeof(int)); moves[0] = 0;
	
	int *g_cuda, *d_cuda, *s_cuda,
		*miscbool_cuda,
		*p_cuda, *a1_cuda, *a2_cuda, *a3_cuda, *a4_cuda, *a5_cuda, *a6_cuda,
		*cores_cuda, *moves_cuda;
	
	hipMalloc(&g_cuda, g[order+1] * sizeof(int));
	hipMalloc(&d_cuda, order * sizeof(int));
	hipMalloc(&s_cuda, order * sizeof(int));
	hipMalloc(&miscbool_cuda, 4 * sizeof(int));
	hipMalloc(&p_cuda, order * sizeof(int));
	hipMalloc(&a1_cuda, order * sizeof(int));
	hipMalloc(&a2_cuda, order * sizeof(int));
	hipMalloc(&a3_cuda, order * sizeof(int));
	hipMalloc(&a4_cuda, order * sizeof(int));
	hipMalloc(&a5_cuda, order * sizeof(int));
	hipMalloc(&a6_cuda, order * sizeof(int));
	hipMalloc(&cores_cuda, sizeof(int));
	hipMalloc(&moves_cuda, sizeof(int));
	
	hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
	
	hipMemcpy(g_cuda, g, g[order+1] * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_cuda, d, order * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(s_cuda, s, order * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(miscbool_cuda, miscbool, 4 * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(p_cuda, p, order * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(a1_cuda, a1, order * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(a2_cuda, a2, order * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(a3_cuda, a3, order * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(a4_cuda, a4, order * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(a5_cuda, a5, order * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(a5_cuda, a6, order * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cores_cuda, cores, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(moves_cuda, moves, sizeof(int), hipMemcpyHostToDevice);
	
	printf("graph read.\nmemories allocated.\ngraph order: %d\ngraph array size: %d\nstarting execution now.\n", order, g[order+1]);
	
	hipEventRecord(start);
	
	if(op == 1) rungk<<<(order + 1 + 255)/256, 256>>>(g_cuda, order, s_cuda, d_cuda, miscbool_cuda, a1_cuda, a4_cuda, a5_cuda, cores_cuda, moves_cuda);
	else if(op == 2) runhea<<<(order + 2 + 255)/256, 256>>>(g_cuda, order, s_cuda, d_cuda, miscbool_cuda, p_cuda, a1_cuda, a4_cuda, a5_cuda, a6_cuda, cores_cuda, moves_cuda);
	else if(op == 3) runt<<<(order + 2 + 255)/256, 256>>>(g_cuda, order, s_cuda, d_cuda, miscbool_cuda, a1_cuda, a4_cuda, a5_cuda, a6_cuda, cores_cuda, moves_cuda);
	else runlg<<<(order + 2 + 255)/256, 256>>>(g_cuda, order, s_cuda, d_cuda, miscbool_cuda, a1_cuda, a2_cuda, a3_cuda, cores_cuda, moves_cuda);
	
	//run<<<1, order + 1>>>(g_cuda, order, s_cuda, d_cuda, miscbool_cuda, ice_cuda, te_cuda, we_cuda, cores_cuda, moves_cuda);
	//run<<<blocks, number of cores per block>>>(arguments);
	
	hipEventRecord(stop);
	
	hipError_t err = hipGetLastError();
	if ( err != hipSuccess ) {
		std::cout << "\nCUDA Error: " << hipGetErrorString(err) << "\n\n";
		return true;
	}
	
	std::cout << "Execution Completed.\n";
	
	hipMemcpy(miscbool, miscbool_cuda, 4 * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(cores, cores_cuda, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(g, g_cuda, g[order+1] * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(s, s_cuda, order * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(a4, a4_cuda, order * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(a5, a5_cuda, order * sizeof(int), hipMemcpyDeviceToHost);	
	
	hipEventSynchronize(stop);
	float time_taken = 0;
	hipEventElapsedTime(&time_taken, start, stop);
	
	std::string file = "output/e";
	file.append(i2s(size));
	file.append("o");
	file.append(i2s(order));
	file.append("/");
	
	struct stat statFile = {0};
	char* fileCA = s2ca(file);
	if (stat(fileCA, &statFile) == -1) mkdir(fileCA, 0700);
	
	if(op == 1) file.append("gk");
	else if(op == 2) file.append("hea");
	else if(op == 3) file.append("t");
	else file = file.append("lg");
	
	file.append(i2s(order));
	std::ofstream ofs(file);
	
	ofs << "time diff = " << time_taken << "\n";
	std::cout << "time diff = " << time_taken << "\n";
	
	if(miscbool[0] == 0) {
		if(miscbool[1] == 0) {
			std::cout<< "False\n";
			ofs<<"False\n";
		} else {
			std::cout<< "True\n";
			ofs<<"True\n";
		}
	} else {
		std::cout<< "Null\n";
		ofs<<"Null\n";
	}
	printf("cores = %d\n", cores[0]);
	ofs<<"cores = " << cores[0] << "\n";
	
	if(finalCheckMaximallyIndependent(g, s, d, order)) printf("Maximally Independent.\n");
	else printf("Not Maximally Independent.\n");
	
	long int SC = arraySum(a4, order);
	long int SNC = arraySum(a5, order);
	ofs << "state changed " << SC << "\n";
	ofs << "state not changed " << SNC << "\n";
	std::cout << "state changed " << SC << "\n";
	std::cout << "state not changed " << SNC << "\n";
	
	if(op == 2 || op == 3) {
		std::cout << "total locksteps = " << miscbool[3];
		ofs << "total locksteps = " << miscbool[3];
	}
	
	ofs.close();
	
	hipFree(g_cuda);
	hipFree(d_cuda);
	hipFree(s_cuda);
	hipFree(miscbool_cuda);
	hipFree(p_cuda);
	hipFree(a1_cuda);
	hipFree(a2_cuda);
	hipFree(a3_cuda);
	hipFree(a4_cuda);
	hipFree(a5_cuda);
	hipFree(a6_cuda);
	hipFree(cores_cuda);
	hipFree(moves_cuda);
	
	//total size  = 11 * addr + (11 * order + 2 * edges + 2) * sizeof(int)
	
	free(miscbool);
	free(p);
	free(a1);
	free(a2);
	free(a3);
	free(a4);
	free(a5);
	free(a6);
	free(cores);
	free(moves);
	
	return false;
}

int main(void)
{
	printf("\nHi ATG.! Please wait while I read the graph...\n");
	GraphManager* gm = new GraphManager();
	//IntegerSet **is = gm->readGraphGenericStyle("graphs/out.hiv"); //40,1,nodes,startingFrom
	IntegerSet **is = gm->readGraphGenericStyle("graphs/lastfm_asia_target.csv"); //7624,0,nodes,startingFrom,27806,edges
	//IntegerSet **is = gm->readGraphGenericStyle("graphs/musae_git_edges.csv"); //37700,0,nodes,startingFrom,289003,edges
	//IntegerSet **is = gm->readGraphGenericStyle("graphs/large_twitch_edges.csv"); //168114,0,nodes,startingFrom,6797557,edges
	//IntegerSet **is = gm->readGraphGenericStyle("graphs/out.roadNet-TX"); //1379917,1,nodes,startingFrom,1921660,edges
	
	int order = gm->getOrder();
	int size = gm->getSize();
	
	printf("Now converting to 1D (plus sorting, so, may take time)...\n");
	
	int* d = gm->getDegrees(is); // degrees
	int *g = gm->graphAdjacencyTo1D(is, d); //graph
	int* s = gm->readStates();
	
	free(gm);
	free(is);
	
	printf("I finished reading the graph. Thanks for waiting, ATG.!\n");
	bool exit = false;
	while(!exit) exit = driver(d, g, s, order, size);
	
	free(g);
	free(d);
	free(s);
}
